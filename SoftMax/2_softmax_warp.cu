#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 32 * 1024 * 1024
#define THREAD_PER_BLOCK 256

// ROWS��block��һ��block��һ�е�max��sum���Լ�softmmax
// 2dims-tensor SoftMax��3��loop
template <int Rows, int Cols>
__global__ void softmaxNomal(float *input, float *output)
{
    uint tid = threadIdx.x;
    uint warpID = tid / WARPSIZE;
    uint laneID = tid % WARPSIZE;
    unit warpSum = blockDim.x / WARPSIZE;

    float vmax = -INFINITY;
    for (int i = tid; i < N; i += blockDim.x)
    {
        val = max(val, input[blockIdx.x * N + i]);
    }
    vmax = BlockAllReduce<float, MaxOp>(val);
    __syncthreads();

    float exp_sum = 1e-10;
    for (int i = tid; i < N; i += blockDim.x)
    {
        exp_sum += __expf((inptu[blockIdx.x * N + i] - vmax) * softmax_scale);
    }
    exp_sum = BlockAllReduce<float, SumOp>(exp_sum);
    __syncthreads();

    for (int i = tid; i < N; i += blockDim.x)
    {
        output[blockIdx.x * N + i] = __expf((inptu[blockIdx.x * N + i] - vmax) * softmax_scale)/exp_sum;
    }
}

template <int Rows, int Cols>
void CpuCalculate(float *input, float *output)
{
    for (int tidx = 0; tidx < Rows; tidx++)
    {
        float *input_start = input + tidx * Cols;
        float *output_start = output + tidx * Cols;

        float max_val = -1e20f;
        for (int i = 0; i < Cols; i++)
        {
            max_val = max(max_val, input_start[i]);
        }

        float sum_val = 0.f;
        for (int i = 0; i < Cols; i++)
        {
            float e = expf(input_start[i] - max_val);
            sum_val += e;
            output_start[i] = e;
        }

        for (int i = 0; i < Cols; i++)
        {
            output_start[i] /= sum_val;
        }
    }
}

template <int Rows, int Cols>
bool checkResult(float *cpu_result, float *gpu_result)
{
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
        {
            if (abs(cpu_result[i * Cols + j] - gpu_result[i * Cols + j]) > 0.01)
                return false;
        }
    }
    return true;
}

template <int Rows, int Cols>
void printResult(float *gpu_result)
{
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
        {
            printf("%f ", gpu_result[i * Cols + j]);
        }
        printf("\n");
    }
}

int main()
{
    constexpr int Rows = 256;
    constexpr int Cols = 256;
    // cpu alloc
    float *input = (float *)malloc(Rows * Cols * sizeof(float));
    float *gpu_result = (float *)malloc(Rows * Cols * sizeof(float));

    float *cpu_result = (float *)malloc(Rows * Cols * sizeof(float));

    // init
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
            input[i * Cols + j] = j;
    }

    // cpu result
    CpuCalculate<Rows, Cols>(input, cpu_result);

    // gpu alloc
    float *d_input;
    hipMalloc((void **)&d_input, Rows * Cols * sizeof(float));
    hipMemcpy(d_input, input, Rows * Cols * sizeof(float), hipMemcpyHostToDevice);

    float *d_output = (float *)malloc(Rows * Cols * sizeof(float));
    hipMalloc((void **)&d_output, Rows * Cols * sizeof(float));

    dim3 Grid(1, 1);
    dim3 Block(Rows, 1);
    softmaxNomal<Rows, Cols><<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(gpu_result, d_output, Rows * Cols * sizeof(float), hipMemcpyDeviceToHost);

    if (checkResult<Rows, Cols>(cpu_result, gpu_result))
    {
        printf("Result is correct!\n");
        // printResult<Rows, Cols>(gpu_result);
    }
    else
        printf("Result is incorret!\n");
}