#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 32 * 1024 * 1024
#define THREAD_PER_BLOCK 256
#define WARPSIZE 32

// gridDim: Rows
// blockDim: blockDim.x, �����32�ı�����WARPSIZE��

// һ��block���һ�ε��
// input: A: M x N, b: N
// output: c : N
__global__ void gemvNormal(float *A, float *b, float *c, int M, int N)
{
    uint tid = threadIdx.x;
    uint warpID = tid/WARPSIZE;
    uint laneID = tid%WARPSIZE;
    unit warpSum = blockDim.x / WARPSIZE;

    float val = 0.f;
    for(int i = tid; i < N; i += blockDim.x)
    {
        val += A[blockIdx.x * N + i] * b[i]
    }

    __shared__ float sum_smem[32];

    for(int i = WARPSIZE >> 2; i > 0; i >>= 1)
    {
      val += __shfl_down_sync(0xffffffff, val, i);
    }

    if(laneID == 0) sum_smem[warpID] = val;

    __syncthreads();
    
    if(warpID == 0)
    {
      float val = laneID < warpSum ? sum_smem[laneID] : 0.f;
      for(int i = WARPSIZE >> 2; i > 0; i >>= 1)
      {
        val += __shfl_down_sync(0xffffffff, val, i);
      }
      if(laneID == 0) c[0] = val;
    }
}

template <int Rows, int Cols>
void CpuCalculate(float *input, float *output)
{
    for (int tidx = 0; tidx < Rows; tidx++)
    {
        float *input_start = input + tidx * Cols;
        float *output_start = output + tidx * Cols;

        float max_val = -1e20f;
        for (int i = 0; i < Cols; i++)
        {
            max_val = max(max_val, input_start[i]);
        }

        float sum_val = 0.f;
        for (int i = 0; i < Cols; i++)
        {
            float e = expf(input_start[i] - max_val);
            sum_val += e;
            output_start[i] = e;
        }

        for (int i = 0; i < Cols; i++)
        {
            output_start[i] /= sum_val;
        }
    }
}

template <int Rows, int Cols>
bool checkResult(float *cpu_result, float *gpu_result)
{
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
        {
            if (abs(cpu_result[i * Cols + j] - gpu_result[i * Cols + j]) > 0.01)
                return false;
        }
    }
    return true;
}

template <int Rows, int Cols>
void printResult(float *gpu_result)
{
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
        {
            printf("%f ", gpu_result[i * Cols + j]);
        }
        printf("\n");
    }
}

int main()
{
    constexpr int Rows = 256;
    constexpr int Cols = 256;
    // cpu alloc
    float *input = (float *)malloc(Rows * Cols * sizeof(float));
    float *gpu_result = (float *)malloc(Rows * Cols * sizeof(float));

    float *cpu_result = (float *)malloc(Rows * Cols * sizeof(float));

    // init
    for (int i = 0; i < Rows; i++)
    {
        for (int j = 0; j < Cols; j++)
            input[i * Cols + j] = j;
    }

    // cpu result
    CpuCalculate<Rows, Cols>(input, cpu_result);

    // gpu alloc
    float *d_input;
    hipMalloc((void **)&d_input, Rows * Cols * sizeof(float));
    hipMemcpy(d_input, input, Rows * Cols * sizeof(float), hipMemcpyHostToDevice);

    float *d_output = (float *)malloc(Rows * Cols * sizeof(float));
    hipMalloc((void **)&d_output, Rows * Cols * sizeof(float));

    dim3 Grid(1, 1);
    dim3 Block(Rows, 1);
    softmaxNomal<<<Grid, Block>>>(d_input, d_output, Rows, Cols);

    hipMemcpy(gpu_result, d_output, Rows * Cols * sizeof(float), hipMemcpyDeviceToHost);

    if (checkResult<Rows, Cols>(cpu_result, gpu_result))
    {
        printf("Result is correct!\n");
        // printResult<Rows, Cols>(gpu_result);
    }
    else
        printf("Result is incorret!\n");
}