#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define N 32*1024*1024
#define THREAD_PER_BLOCK 256

__global__ void reduceNomal(float *d_input, float *d_output)
{
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int thread_id = threadIdx.x;
    unsigned int global_id = threadIdx.x + blockDim.x*blockIdx.x;
    sdata[thread_id] = d_input[global_id];
    __syncthreads();

    for(int i = 1; i < blockDim.x; i *= 2)
    {
        if(thread_id % (i*2) == 0)
            sdata[thread_id] += sdata[thread_id + i];
        __syncthreads();
    } 
    if(thread_id == 0)
        d_output[blockIdx.x] = sdata[0];
}

bool checkResult(float *out, float *res, int n)
{
    for(int i = 0; i < n; i++)
    {
        if(abs(out[i] - res[i]) > 0.5) return false;
    }
    return true;
}

int main()
{
    // cpu malloc
    float *input = (float *)malloc(N*sizeof(float));
    int num_block = ceil((float)N/THREAD_PER_BLOCK);
    float *output = (float *)malloc(num_block*sizeof(float));

    float *cpu_result = (float *)malloc(num_block*sizeof(float));

    // init
    for(int i = 0; i < N; i++)
    {
        input[i] = 1; 
    }

    // cpu result
    for(int i = 0; i < num_block; i++)
    {
        float ans = .0;
        for(int j = 0; j < ((i < num_block - 1)? 
                            THREAD_PER_BLOCK:
                            N - (num_block - 1)*THREAD_PER_BLOCK); j++)
        {
            ans += input[i*THREAD_PER_BLOCK + j];
        }
        
        cpu_result[i] = ans;
    }

    // gpu malloc
    float *d_input;
    hipMalloc((void **)&d_input, N*sizeof(float));
    hipMemcpy(d_input, input, N*sizeof(float), hipMemcpyHostToDevice);
    
    float *d_output = (float *)malloc(num_block*sizeof(float));
    hipMalloc((void **)&d_output, num_block*sizeof(float));

    dim3 Grid(num_block, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    reduceNomal<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, num_block*sizeof(float), hipMemcpyDeviceToHost);
    
    if(checkResult(output, cpu_result, num_block)) 
        printf("Result is correct!\n");
    else 
        printf("Result is incorret!\n");

}